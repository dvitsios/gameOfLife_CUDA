#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2010 by Vitsios Dimitrios
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy
 *  of this software and associated documentation files (the "Software"), to deal
 *  in the Software without restriction, including without limitation the rights
 *  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *  copies of the Software, and to permit persons to whom the Software is
 *  furnished to do so, subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in
 *  all copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *  THE SOFTWARE.
 */

/*************************************************
*						 *   
*  Description: a "game of life" implementation  *
*						 *				    
*		        ~ using CUDA ~  	 *
*						 *            
*************************************************/


#include <stdio.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLOCK_SIZE 256


char *host_board;
int  n, t;



__global__ void make_move(char *dev_board, int n)
{
           
		   __shared__ char block_brd[324], int sum[256], flags[256];
		   int i, j;
		   int local_Idx  = threadIdx.x + threadIdx.y * 16; 
		   flags[local_Idx]=0;
		   
		  
           
           int ix = blockIdx.x * blockDim.x + threadIdx.x;
           int iy = blockIdx.y * blockDim.y + threadIdx.y;
           int index = ix + iy * n;
           
	       int th_idx=threadIdx.x + 1 + (threadIdx.y + 1) * 18;
           
           block_brd[th_idx] = dev_board[index]; //fill in (a part of) the matrix 'block board' with the '256' elements to process 
           
           
		   //fill in with the remaining '68' elements
           
           int ix_ul = blockIdx.x * blockDim.x;  //coordinates for the upper left corner of the quadratic table...
           int iy_ul = blockIdx.y * blockDim.y;  //...(size 16x16) containing the 256 elements designated for processing   
           int upperLeftCorner = (ix_ul == 0 && iy_ul == 0)? (n*n-1): ix_ul - 1 + (iy_ul - 1)*n;
           block_brd[0] = dev_board[upperLeftCorner];
           
           int ix_ur = ix_ul+15;  //coordinates for the upper right corner...
           int iy_ur = iy_ul;	  //...of the quadratic table 
           int upperRightCorner = (ix_ur == n-1 && iy_ur == 0)? n*(n-1): ix_ur + 1 + (iy_ur - 1)*n; 
           block_brd[17] = dev_board[upperRightCorner];
           
           int ix_bl = ix_ul;     //coordinates for the bottom left corner...
           int iy_bl = iy_ul+15;  //...of the quadratic table
           int bottomLeftCorner = (ix_bl == 0 && iy_bl == n-1)? n-1: ix_bl - 1 + (iy_bl + 1)*n;
           block_brd[306] = dev_board[bottomLeftCorner];
           
           int ix_br = ix_ul+15;   //coordinates for the bottom right corner...
           int iy_br = iy_ul+15;   //...of the quadratic table
           int bottomRightCorner = (ix_br == n-1 && iy_br == n-1)? 0: ix_br + 1 + (iy_br + 1)*n; 
           block_brd[323] = dev_board[bottomRightCorner];
           
           
           //Upper Row
           for(int k=0; k<16;k++){
				
				int urIdx1 = (iy_ul == 0)? n*(n-1)+ix_ul+k: (ix_ul+k) + (iy_ul-1) * n;		
				
				block_brd[k+1] = dev_board[urIdx1];
		   }		
           
           //Right Column
           for(int k=0, i=35; k<16; k++, i+=18){
				
				int urIdx2 = (ix_ur == n-1)? n*(iy_ul+k):(ix_ur+1) + (iy_ul+k) * n;		
				
				block_brd[i] = dev_board[urIdx2];
		   }
           
           //Bottom Row
           for(int k=0, i=307; k<16;k++, i++){
				
				int urIdx3 = (iy_bl == n-1)? ix_ul+k: (ix_bl+k) + (iy_bl+1) * n;		
				
				block_brd[i] = dev_board[urIdx3];
		   }
		   
		   //Left Column
           for(int k=0, i=18; k<16;k++, i+=18){
				
				int urIdx4 = (ix_ul == 0)? n*(iy_ul+1+k)-1 :(ix_ul-1) + (iy_ul+k) * n;		
				
				block_brd[i] = dev_board[urIdx4];
		   } 
		   
		   
           if ( index < n*n ){
				
			 
           
           
              sum[local_Idx]   =	  (block_brd[threadIdx.x + threadIdx.y * 18])
									 +(block_brd[threadIdx.x + 1 + threadIdx.y * 18])
								     +(block_brd[threadIdx.x + 2 + threadIdx.y * 18])
								     +(block_brd[threadIdx.x + (threadIdx.y + 1) * 18])
								     +(block_brd[threadIdx.x + 2 + (threadIdx.y + 1) * 18])
								     +(block_brd[threadIdx.x + (threadIdx.y + 2) * 18])
								     +(block_brd[threadIdx.x + 1 + (threadIdx.y + 2) * 18])
								     +(block_brd[threadIdx.x + 2 + (threadIdx.y + 2) * 18]);
						 
						 
						 
           if(block_brd[th_idx]==0 && sum[local_Idx]==3)
               flags[local_Idx]=1;
           if(block_brd[th_idx]==1 && (sum[local_Idx]<2 || sum[local_Idx]>3))
               flags[local_Idx]=2;
                  
           __syncthreads();
		   
           if(flags[local_Idx] == 1)
	       	dev_board[index]=1;
           
		   if(flags[local_Idx] == 2)
            dev_board[index]=0;
			
           }
	
}




int main(int argc, char* argv[]){
    
	FILE *Data_File;
	
	int *br,i,j;
	char inFile[256], *inFileName=inFile, test_ch, outFileName[256], *dev_board;;
	int ncount=0;
	time_t start, end;
	
	if (argc != 3 && argc !=1) {
		printf("Insufficient parameters!\n");
		exit(1);
	}
	else{
		if (argc == 1){
			printf("Type the number of iterations: ");
			scanf("%d",&t);
			printf("\nType the name of the data file: ");
			scanf("%s",inFile);
			printf("\n\n");
		}
		else{
				t=atoi(argv[1]);
				inFileName=argv[2];
		}
	}
	Data_File=fopen(inFileName,"r");

	do{
		fscanf(Data_File, "%c", &test_ch);
		ncount++;
	}while(test_ch!='\n');
	n=ncount/2;   // in d.txt: ncounter = 600 --> 300, the numbers (0,1) and 300, the spaces. So: n=ncounter/2 
	fseek(Data_File,0,SEEK_SET);

	
	int size = n * n *sizeof(char);
	host_board=(char *)malloc(size);
	
	for(i=0;i<n;i++){
	        for(j=0;j<n;j++){
	                fscanf(Data_File,"%c ",&host_board[i+j*n]);
					host_board[i+j*n]-=48;
	        }
	}
	                
    printf("Reading done\n\n");
	fclose(Data_File);
	
	//Start timer...
    time(&start);
    
	hipMalloc((void**)&dev_board,size);

	hipMemcpy( dev_board, host_board, size, hipMemcpyHostToDevice );
	
	printf("Transfer done\n\n");
	
	
    dim3 dimBlock(16,16);
    dim3 dimGrid( (n/dimBlock.x) , (n/dimBlock.y) );


    for(int r=0; r<t; r++)
    {       

			make_move<<< dimGrid, dimBlock>>>(dev_board, n);
			
    }
    
    hipMemcpy(host_board, dev_board, size, hipMemcpyDeviceToHost);
    
    printf("GPU PROCESSING COMPLETE!\n\n");

	//Stop timer;
    time(&end);
    
    
	//Writing to the output data file
	i=0;
	do{
		outFileName[i]=inFileName[i];
		i++;
	}while(inFileName[i]!=0);
	outFileName[i]='.';
	outFileName[i+1]='o';
	outFileName[i+2]='u';
	outFileName[i+3]='t';
	outFileName[i+4]=0;
	printf("Output File \''%s\'' was created!\n",outFileName);
	Data_File=fopen(outFileName,"w");

	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			fprintf(Data_File,"%i",host_board[i+j*n]);
			if(j!=n-1)
				fprintf(Data_File," ");
		}
		if(i!=n-1)
			fprintf(Data_File,"\n");
	}
    hipFree(dev_board);
    free(host_board);
    
    double dif=difftime(end,start);
    printf("\n*******************************************************************************");
    printf("\nTotal time elapsed for transfering the data and computing in GPU: %.2lf seconds",dif);
    
    scanf("%d",&i);
    return EXIT_SUCCESS;
}


